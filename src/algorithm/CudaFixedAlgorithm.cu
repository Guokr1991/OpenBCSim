#include "hip/hip_runtime.h"
/*
Copyright (c) 2015, Sigurd Storve
All rights reserved.

Licensed under the BSD license.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of the <organization> nor the
      names of its contributors may be used to endorse or promote products
      derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <complex>
#include "CudaFixedAlgorithm.cuh"
#include "cuda_helpers.h"
#include "cufft_helpers.h"
#include "" // for removing annoying MSVC intellisense error messages
#include "discrete_hilbert_mask.hpp"
#include "common_utils.hpp" // for compute_num_rf_samples
#include "gpu_alg_common.cuh" // for misc. CUDA kernels

namespace bcsim {


__global__ void FixedAlgKernel(float* point_xs,
                               float* point_ys,
                               float* point_zs,
                               float* point_as,
                               float3 rad_dir,
                               float3 lat_dir,
                               float3 ele_dir,
                               float3 origin,
                               float  fs_hertz,
                               int    num_time_samples,
                               float  sigma_lateral,
                               float  sigma_elevational,
                               float  sound_speed,
                               float* res) {

    const int global_idx = blockIdx.x*blockDim.x + threadIdx.x;

    float3 point = make_float3(point_xs[global_idx], point_ys[global_idx], point_zs[global_idx]) - origin;
    
    // compute dot products
    const auto radial_dist  = dot(point, rad_dir);
    const auto lateral_dist = dot(point, lat_dir);
    const auto elev_dist    = dot(point, ele_dir);


    const float two_sigma_lateral_squared     = 2.0f*sigma_lateral*sigma_lateral;
    const float two_sigma_elevational_squared = 2.0f*sigma_elevational*sigma_elevational; 
    const float weight = expf(-(lateral_dist*lateral_dist/two_sigma_lateral_squared + elev_dist*elev_dist/two_sigma_elevational_squared));

    const int radial_index = static_cast<int>(fs_hertz*2.0f*radial_dist/sound_speed + 0.5f);
    
    if (radial_index >= 0 && radial_index < num_time_samples) {
        //res[radial_index] += weight;
        atomicAdd(res+radial_index, weight*point_as[global_idx]);
    }
}


CudaFixedAlgorithm::CudaFixedAlgorithm()
    : m_verbose(false),
      m_num_cuda_streams(2),
      m_num_time_samples(32768),  // TODO: remove this limitation
      m_num_beams_allocated(-1),
      m_beam_profile(nullptr),
      m_output_type("env")
{
}

void CudaFixedAlgorithm::simulate_lines(std::vector<std::vector<bc_float> >&  /*out*/ rf_lines) {
    cudaErrorCheck(hipSetDevice(m_cuda_device_no));
    m_can_change_cuda_device = false;
    
    if (m_stream_wrappers.size() == 0) {
        create_cuda_stream_wrappers(m_num_cuda_streams);
    }
    
    auto num_lines      = m_scan_seq->get_num_lines();

    if (num_lines < 1) {
        throw std::runtime_error("No scanlines in scansequence");
    }

    if (m_beam_profile == nullptr) {
        throw std::runtime_error("No beam profile is set");
    }
    
    int threads_pr_block = 128;
    dim3 grid_size(m_num_scatterers/threads_pr_block, 1, 1);
    dim3 block_size(threads_pr_block, 1, 1);

    for (int beam_no = 0; beam_no < num_lines; beam_no++) {
        size_t stream_no = beam_no % m_num_cuda_streams;
        auto cur_stream = m_stream_wrappers[stream_no]->get();

        if (m_verbose) {
            std::cout << "beam_no = " << beam_no << ", stream_no = " << stream_no << std::endl;
        }

        // TODO: Move out conversion code            
        auto scanline = m_scan_seq->get_scanline(beam_no);
        auto temp_rad_dir = scanline.get_direction();
        auto temp_lat_dir = scanline.get_lateral_dir();
        auto temp_ele_dir = scanline.get_elevational_dir();
        auto temp_origin  = scanline.get_origin();
        auto rad_dir      = make_float3(temp_rad_dir.x, temp_rad_dir.y, temp_rad_dir.z);
        auto lat_dir      = make_float3(temp_lat_dir.x, temp_lat_dir.y, temp_lat_dir.z);
        auto ele_dir      = make_float3(temp_ele_dir.x, temp_ele_dir.y, temp_ele_dir.z);
        auto origin       = make_float3(temp_origin.x, temp_origin.y, temp_origin.z);

        //std::cout << "origin: " << origin.x << " " << origin.y << " " << origin.z << std::endl;

        int threads_per_line = 128;
        // clear the time projection buffer the proper way (probably slightly slower than cudaMamSetAsync...)
        MemsetFloatKernel<<<m_num_time_samples/threads_per_line, threads_per_line, 0, cur_stream>>>(m_device_time_proj[stream_no]->data(),
                                                                                                    0.0f,
                                                                                                    m_num_time_samples);

        //if (beam_no==0) { dump_device_memory<float>(device_time_proj[stream_no]->data(), m_num_time_samples, "01_zeroed_rf_line_dump.txt"); }
        
        // do the time-projections
        FixedAlgKernel<<<grid_size, block_size, 0, cur_stream>>>(m_device_point_xs->data(),
                                                                 m_device_point_ys->data(),
                                                                 m_device_point_zs->data(),
                                                                 m_device_point_as->data(),
                                                                 rad_dir,
                                                                 lat_dir,
                                                                 ele_dir,
                                                                 origin,
                                                                 m_excitation.sampling_frequency,
                                                                 m_num_time_samples,
                                                                 m_beam_profile->getSigmaLateral(),
                                                                 m_beam_profile->getSigmaElevational(),
                                                                 m_sound_speed,
                                                                 m_device_time_proj[stream_no]->data());
            
        //if (beam_no==0) { dump_device_memory<float>(device_time_proj[stream_no]->data(), m_num_time_samples, "02_time_proj_dump.txt"); }


        // extend the real-valued time-projection signal to complex numbers with zero imaginary part.
        RealToComplexKernel<<<m_num_time_samples/threads_per_line, threads_per_line, 0, cur_stream>>>(m_device_time_proj[stream_no]->data(),
                                                                                                      m_device_rf_lines[stream_no]->data(),
                                                                                                      m_num_time_samples);
        //if (beam_no==0) { dump_device_memory<std::complex<float> >(reinterpret_cast<std::complex<float>*>(device_rf_lines[stream_no]->data()), m_num_time_samples, "03_complex_extension.txt"); }

        // in-place forward FFT            
        auto rf_ptr = m_device_rf_lines[stream_no]->data();
        cufftErrorCheck( hipfftExecC2C(m_fft_plan->get(), rf_ptr, rf_ptr, HIPFFT_FORWARD) );

        // multiply with FFT of impulse response (can include Hilbert transform also)
        MultiplyFftKernel<<<m_num_time_samples/threads_per_line, threads_per_line, 0, cur_stream>>>(m_device_rf_lines[stream_no]->data(),
                                                                                                    m_device_excitation_fft->data(),
                                                                                                    m_num_time_samples);

        // in-place inverse FFT
        cufftErrorCheck( hipfftExecC2C(m_fft_plan->get(), rf_ptr, rf_ptr, HIPFFT_BACKWARD) );
            
        //if (beam_no==0) { dump_device_memory<std::complex<float> >(reinterpret_cast<std::complex<float>*>(rf_ptr), m_num_time_samples, "04_iq_line.txt"); }

        if (m_output_type == "env") {
            // envelope detection
            AbsComplexKernel<<<m_num_time_samples/threads_per_line, threads_per_line, 0, cur_stream>>>(m_device_rf_lines[stream_no]->data(),
                                                                                                    m_device_rf_lines_env[stream_no]->data(),
                                                                                                    m_num_time_samples);
        } else if (m_output_type == "rf") {
            // rf data
            RealPartKernel<<<m_num_time_samples/threads_per_line, threads_per_line, 0, cur_stream>>>(m_device_rf_lines[stream_no]->data(),
                                                                                                     m_device_rf_lines_env[stream_no]->data(),
                                                                                                     m_num_time_samples);
            
        } else {
            throw std::logic_error("illegal output type");        
        }
        //if (beam_no==0) { dump_device_memory<float>(device_rf_lines_env[stream_no]->data(), m_num_time_samples, "05_rf_envelope.txt"); }
            
        // copy to host
        cudaErrorCheck( hipMemcpyAsync(m_host_rf_lines[beam_no]->data(), m_device_rf_lines_env[stream_no]->data(), sizeof(float)*m_num_time_samples, hipMemcpyDeviceToHost, cur_stream) ); 
    }
    cudaErrorCheck( hipDeviceSynchronize() );

    // TODO: eliminate unneccessary data copying: it would e.g. be better to
    // only copy what is needed in the above kernel.
    
    const auto num_return_samples = compute_num_rf_samples(m_sound_speed, m_scan_seq->line_length, m_excitation.sampling_frequency);

    // compensate for delay
    const size_t start_idx = static_cast<size_t>(m_excitation.center_index);

    rf_lines.clear();
    std::vector<bc_float> temp_samples(num_return_samples);
    for (size_t line_no = 0; line_no < num_lines; line_no++) {
        for (size_t i = 0; i < num_return_samples; i++) {
            temp_samples[i] = m_host_rf_lines[line_no]->data()[i+start_idx];
        }
        rf_lines.push_back(temp_samples);
    }
}

void CudaFixedAlgorithm::copy_scatterers_to_device(FixedScatterers::s_ptr scatterers) {
    m_can_change_cuda_device = false;
    
    const size_t num_scatterers = scatterers->num_scatterers();
    size_t points_common_bytes = num_scatterers*sizeof(float);

    // temporary host memory for scatterer points
    HostPinnedBufferRAII<float> host_temp(points_common_bytes);

    // no point in reallocating if we already have allocated memory and the number of bytes
    // is the same.
    bool reallocate_device_mem = true;
    if (m_device_point_xs && m_device_point_ys && m_device_point_zs && m_device_point_as) {
        if (   (m_device_point_xs->get_num_bytes() == points_common_bytes)
            && (m_device_point_ys->get_num_bytes() == points_common_bytes)
            && (m_device_point_zs->get_num_bytes() == points_common_bytes)
            && (m_device_point_as->get_num_bytes() == points_common_bytes))
        {
            reallocate_device_mem = false;
        }
    }
    if (reallocate_device_mem) {
        m_device_point_xs = std::move(DeviceBufferRAII<float>::u_ptr(new DeviceBufferRAII<float>(points_common_bytes)));
        m_device_point_ys = std::move(DeviceBufferRAII<float>::u_ptr(new DeviceBufferRAII<float>(points_common_bytes)));
        m_device_point_zs = std::move(DeviceBufferRAII<float>::u_ptr(new DeviceBufferRAII<float>(points_common_bytes)));
        m_device_point_as = std::move(DeviceBufferRAII<float>::u_ptr(new DeviceBufferRAII<float>(points_common_bytes)));
    }

    // x values
    for (size_t i = 0; i < num_scatterers; i++) {
        host_temp.data()[i] = scatterers->scatterers[i].pos.x;
    }
    cudaErrorCheck( hipMemcpy(m_device_point_xs->data(), host_temp.data(), points_common_bytes, hipMemcpyHostToDevice) );

    // y values
    for (size_t i = 0; i < num_scatterers; i++) {
        host_temp.data()[i] = scatterers->scatterers[i].pos.y;
    }
    cudaErrorCheck( hipMemcpy(m_device_point_ys->data(), host_temp.data(), points_common_bytes, hipMemcpyHostToDevice) );

    // z values
    for (size_t i = 0; i < num_scatterers; i++) {
        host_temp.data()[i] = scatterers->scatterers[i].pos.z;
    }
    cudaErrorCheck( hipMemcpy(m_device_point_zs->data(), host_temp.data(), points_common_bytes, hipMemcpyHostToDevice) );

    // a values
    for (size_t i = 0; i < num_scatterers; i++) {
        host_temp.data()[i] = scatterers->scatterers[i].amplitude;
    }
    cudaErrorCheck( hipMemcpy(m_device_point_as->data(), host_temp.data(), points_common_bytes, hipMemcpyHostToDevice) );
}

void CudaFixedAlgorithm::set_excitation(const ExcitationSignal& new_excitation) {
    m_can_change_cuda_device = false;
    
    m_excitation = new_excitation;
    size_t rf_line_bytes   = sizeof(complex)*m_num_time_samples;

    // setup pre-computed convolution kernel and Hilbert transformer.
    m_device_excitation_fft = DeviceBufferRAII<complex>::u_ptr(new DeviceBufferRAII<complex>(rf_line_bytes));
    std::cout << "Number of excitation samples: " << m_excitation.samples.size() << std::endl;
    // convert to complex with zero imaginary part.
    std::vector<std::complex<float> > temp(m_num_time_samples);
    for (size_t i = 0; i < m_excitation.samples.size(); i++) {
        temp[i] = std::complex<float>(m_excitation.samples[i], 0.0f);
    }
    cudaErrorCheck( hipMemcpy(m_device_excitation_fft->data(), temp.data(), rf_line_bytes, hipMemcpyHostToDevice) );
    //dump_device_memory((std::complex<float>*)m_device_excitation_fft.data(), m_num_time_samples, "complex_exitation.txt");

    m_fft_plan = CufftPlanRAII::u_ptr(new CufftPlanRAII(m_num_time_samples, HIPFFT_C2C, 1));

    // compute FFT of excitation signal and add the Hilbert transform
    cufftErrorCheck( hipfftExecC2C(m_fft_plan->get(), m_device_excitation_fft->data(), m_device_excitation_fft->data(), HIPFFT_FORWARD) );
    auto mask = discrete_hilbert_mask<std::complex<float> >(m_num_time_samples);
    DeviceBufferRAII<complex> device_hilbert_mask(rf_line_bytes);
    cudaErrorCheck( hipMemcpy(device_hilbert_mask.data(), mask.data(), rf_line_bytes, hipMemcpyHostToDevice) );
    
    ScaleSignalKernel<<<m_num_time_samples/128, 128>>>(m_device_excitation_fft->data(), 1.0f/m_num_time_samples, m_num_time_samples);
    
    if (m_output_type == "env") {
        MultiplyFftKernel<<<m_num_time_samples/128, 128>>>(m_device_excitation_fft->data(), device_hilbert_mask.data(), m_num_time_samples);
    }
    //dump_device_memory((std::complex<float>*) m_device_excitation_fft->data(), m_num_time_samples, "complex_excitation_fft.txt");
}


void CudaFixedAlgorithm::set_scan_sequence(ScanSequence::s_ptr new_scan_sequence) {
    m_can_change_cuda_device = false;
    m_scan_seq = new_scan_sequence;

    // HACK: Temporarily limited to the hardcoded value for m_num_time_samples
    auto num_rf_samples = compute_num_rf_samples(m_sound_speed, m_scan_seq->line_length, m_excitation.sampling_frequency);
    //std::cout << "num_rf_samples: " << num_rf_samples << std::endl;
    if (num_rf_samples > m_num_time_samples) {
        throw std::runtime_error("Too many RF samples required. TODO: remove limitation");
    }

    size_t num_beams = m_scan_seq->get_num_lines();
    // avoid reallocating memory if not necessary.
    if (m_num_beams_allocated < static_cast<int>(num_beams)) {
        std::cout << "Allocating HOST and DEVICE memory: had previously allocated memory for " << m_num_beams_allocated << " beams.\n";
    } else {
        return;
    }

    // allocate host and device memory related to RF lines
    size_t time_proj_bytes = sizeof(float)*m_num_time_samples;
    size_t rf_line_bytes   = sizeof(complex)*m_num_time_samples;
    m_device_time_proj.resize(m_num_cuda_streams);
    m_device_rf_lines.resize(m_num_cuda_streams);
    m_device_rf_lines_env.resize(m_num_cuda_streams);
    for (size_t i = 0; i < m_num_cuda_streams; i++) {
        m_device_time_proj[i]    = std::move(DeviceBufferRAII<float>::u_ptr   ( new DeviceBufferRAII<float>(time_proj_bytes)) ); 
        m_device_rf_lines[i]     = std::move(DeviceBufferRAII<complex>::u_ptr ( new DeviceBufferRAII<complex>(rf_line_bytes)) );
        m_device_rf_lines_env[i] = std::move(DeviceBufferRAII<float>::u_ptr   ( new DeviceBufferRAII<float>(time_proj_bytes)) ); 
    }

    // allocate host memory for all RF lines
    m_host_rf_lines.resize(num_beams);
    for (size_t beam_no = 0; beam_no < num_beams; beam_no++) {
        m_host_rf_lines[beam_no] = std::move(HostPinnedBufferRAII<float>::u_ptr( new HostPinnedBufferRAII<float>(time_proj_bytes)) );
    }

    m_num_beams_allocated = static_cast<int>(num_beams);
}

void CudaFixedAlgorithm::set_scatterers(Scatterers::s_ptr new_scatterers) {
    m_can_change_cuda_device = false;
    m_num_scatterers = new_scatterers->num_scatterers();
        
    auto fixed_scatterers = std::dynamic_pointer_cast<FixedScatterers>(new_scatterers);
    if (!fixed_scatterers) {
        throw std::runtime_error("Cast failed");
    }
    copy_scatterers_to_device(fixed_scatterers);
}


}   // end namespace