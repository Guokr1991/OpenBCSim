#include "hip/hip_runtime.h"
/*
Copyright (c) 2015, Sigurd Storve
All rights reserved.

Licensed under the BSD license.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of the <organization> nor the
      names of its contributors may be used to endorse or promote products
      derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <hip/hip_runtime.h>
#include "GpuFixedAlgorithm.cuh"
#include "cuda_helpers.h"
#include "cufft_helpers.h"
#include "" // for removing annoying MSVC intellisense error messages
#include "gpu_alg_common.cuh" // for misc. CUDA kernels
#include <math_functions.h> // for copysignf

namespace bcsim {

// Gaussian analytical beam profile.
__device__ float ComputeWeightAnalytical(float sigma_lateral,
                                         float sigma_elevational,
                                         float radial_dist,
                                         float lateral_dist,
                                         float elev_dist) {
    const float two_sigma_lateral_squared     = 2.0f*sigma_lateral*sigma_lateral;
    const float two_sigma_elevational_squared = 2.0f*sigma_elevational*sigma_elevational; 
    return expf(-(lateral_dist*lateral_dist/two_sigma_lateral_squared + elev_dist*elev_dist/two_sigma_elevational_squared));
}

__global__ void FixedAlgKernel(float* point_xs,
                               float* point_ys,
                               float* point_zs,
                               float* point_as,
                               float3 rad_dir,
                               float3 lat_dir,
                               float3 ele_dir,
                               float3 origin,
                               float  fs_hertz,
                               int    num_time_samples,
                               float  sigma_lateral,
                               float  sigma_elevational,
                               float  sound_speed,
                               hipComplex* res,
                               bool   use_arc_projection,
                               int    num_scatterers,
                               bool   use_phase_delay,
                               float  demod_freq) {

    const int global_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (global_idx >= num_scatterers) {
        return;
    }

    float3 point = make_float3(point_xs[global_idx], point_ys[global_idx], point_zs[global_idx]) - origin;
    
    // compute dot products
    auto radial_dist  = dot(point, rad_dir);
    const auto lateral_dist = dot(point, lat_dir);
    const auto elev_dist    = dot(point, ele_dir);

    if (use_arc_projection) {
        // Use "arc projection" in the radial direction: use length of vector from
        // beam's origin to the scatterer with the same sign as the projection onto
        // the line.
        radial_dist = copysignf(sqrtf(dot(point,point)), radial_dist);
    }

    const float weight = ComputeWeightAnalytical(sigma_lateral, sigma_elevational, radial_dist, lateral_dist, elev_dist);

    const int radial_index = static_cast<int>(fs_hertz*2.0f*radial_dist/sound_speed + 0.5f);
    
    if (radial_index >= 0 && radial_index < num_time_samples) {
        //atomicAdd(res+radial_index, weight*point_as[global_idx]);
        if (use_phase_delay) {
            // handle sub-sample displacement with a complex phase
            const auto true_index = fs_hertz*2.0f*radial_dist/sound_speed;
            const float ss_delay = (radial_index - true_index)/fs_hertz;
            const float complex_phase = 6.283185307179586*demod_freq*ss_delay;

            // exp(i*theta) = cos(theta) + i*sin(theta)
            float sin_value, cos_value;
            sincosf(complex_phase, &sin_value, &cos_value);

            const auto w = weight*point_as[global_idx];
            atomicAdd(&(res[radial_index].x), w*cos_value);
            atomicAdd(&(res[radial_index].y), w*sin_value);
        } else {
            atomicAdd(&(res[radial_index].x), weight*point_as[global_idx]);
        }
    }
}

__global__ void FixedAlgKernel_LUT(float* point_xs,
                                   float* point_ys,
                                   float* point_zs,
                                   float* point_as,
                                   float3 rad_dir,
                                   float3 lat_dir,
                                   float3 ele_dir,
                                   float3 origin,
                                   float  fs_hertz,
                                   int    num_time_samples,
                                   float  sound_speed,
                                   hipComplex* res,
                                   bool   use_arc_projection,
                                   int    num_scatterers,
                                   bool   use_phase_delay,
                                   float  demod_freq,
                                   hipTextureObject_t lut_tex) {

    const int global_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (global_idx >= num_scatterers) {
        return;
    }

    float3 point = make_float3(point_xs[global_idx], point_ys[global_idx], point_zs[global_idx]) - origin;
    
    // compute dot products
    auto radial_dist  = dot(point, rad_dir);
    const auto lateral_dist = dot(point, lat_dir);
    const auto elev_dist    = dot(point, ele_dir);

    if (use_arc_projection) {
        // Use "arc projection" in the radial direction: use length of vector from
        // beam's origin to the scatterer with the same sign as the projection onto
        // the line.
        radial_dist = copysignf(sqrtf(dot(point,point)), radial_dist);
    }

    // TODO: Compute weight from lookup-table and radial_dist, lateral_dist, and elev_dist
    const auto lut_r_min = 0.0f; // HACK: this should be sent as parameters
    const auto lut_r_max = 0.12f;
    const auto lut_l_min = -2e-2f;
    const auto lut_l_max = 2e-2f;
    const auto lut_e_min = -2e-2f;
    const auto lut_e_max = 2e-2;
    
    const auto r_normalized = (radial_dist-lut_r_min)/(lut_r_max-lut_r_min);
    const auto l_normalized = (lateral_dist-lut_l_min)/(lut_l_max-lut_l_min);
    const auto e_normalized = (elev_dist-lut_e_min)/(lut_e_max-lut_e_min);
    const auto weight = tex3D<float>(lut_tex, l_normalized, e_normalized, r_normalized);

    const int radial_index = static_cast<int>(fs_hertz*2.0f*radial_dist/sound_speed + 0.5f);
    
    if (radial_index >= 0 && radial_index < num_time_samples) {
        //atomicAdd(res+radial_index, weight*point_as[global_idx]);
        if (use_phase_delay) {
            // handle sub-sample displacement with a complex phase
            const auto true_index = fs_hertz*2.0f*radial_dist/sound_speed;
            const float ss_delay = (radial_index - true_index)/fs_hertz;
            const float complex_phase = 6.283185307179586*demod_freq*ss_delay;

            // exp(i*theta) = cos(theta) + i*sin(theta)
            float sin_value, cos_value;
            sincosf(complex_phase, &sin_value, &cos_value);

            const auto w = weight*point_as[global_idx];
            atomicAdd(&(res[radial_index].x), w*cos_value);
            atomicAdd(&(res[radial_index].y), w*sin_value);
        } else {
            atomicAdd(&(res[radial_index].x), weight*point_as[global_idx]);
        }
    }
}


GpuFixedAlgorithm::GpuFixedAlgorithm()
{
}

void GpuFixedAlgorithm::projection_kernel(int stream_no, const Scanline& scanline, int num_blocks) {
    auto cur_stream = m_stream_wrappers[stream_no]->get();

    // TODO: Move out conversion code            
    auto temp_rad_dir = scanline.get_direction();
    auto temp_lat_dir = scanline.get_lateral_dir();
    auto temp_ele_dir = scanline.get_elevational_dir();
    auto temp_origin  = scanline.get_origin();
    auto rad_dir      = make_float3(temp_rad_dir.x, temp_rad_dir.y, temp_rad_dir.z);
    auto lat_dir      = make_float3(temp_lat_dir.x, temp_lat_dir.y, temp_lat_dir.z);
    auto ele_dir      = make_float3(temp_ele_dir.x, temp_ele_dir.y, temp_ele_dir.z);
    auto origin       = make_float3(temp_origin.x, temp_origin.y, temp_origin.z);

    dim3 grid_size(num_blocks, 1, 1);
    dim3 block_size(m_param_threads_per_block, 1, 1);
    
    // Use casting of lookup_table to determine which kernel to call
    const auto gaussian_beam_profile = std::dynamic_pointer_cast<bcsim::GaussianBeamProfile>(m_beam_profile);
    const auto lut_beam_profile      = std::dynamic_pointer_cast<bcsim::LUTBeamProfile>(m_beam_profile);

    if (gaussian_beam_profile) {
        FixedAlgKernel<<<grid_size, block_size, 0, cur_stream>>>(m_device_point_xs->data(),
                                                                 m_device_point_ys->data(),
                                                                 m_device_point_zs->data(),
                                                                 m_device_point_as->data(),
                                                                 rad_dir,
                                                                 lat_dir,
                                                                 ele_dir,
                                                                 origin,
                                                                 m_excitation.sampling_frequency,
                                                                 m_num_time_samples,
                                                                 gaussian_beam_profile->getSigmaLateral(),
                                                                 gaussian_beam_profile->getSigmaElevational(),
                                                                 m_param_sound_speed,
                                                                 m_device_time_proj[stream_no]->data(),
                                                                 m_param_use_arc_projection,
                                                                 m_num_scatterers,
                                                                 m_enable_phase_delay,
                                                                 m_excitation.demod_freq);
    } else if (lut_beam_profile) {
        FixedAlgKernel_LUT<<<grid_size, block_size, 0, cur_stream>>>(m_device_point_xs->data(),
                                                                     m_device_point_ys->data(),
                                                                     m_device_point_zs->data(),
                                                                     m_device_point_as->data(),
                                                                     rad_dir,
                                                                     lat_dir,
                                                                     ele_dir,
                                                                     origin,
                                                                     m_excitation.sampling_frequency,
                                                                     m_num_time_samples,
                                                                     m_param_sound_speed,
                                                                     m_device_time_proj[stream_no]->data(),
                                                                     m_param_use_arc_projection,
                                                                     m_num_scatterers,
                                                                     m_enable_phase_delay,
                                                                     m_excitation.demod_freq,
                                                                     m_device_beam_profile->get()
                                                                     );
    }
}


void GpuFixedAlgorithm::copy_scatterers_to_device(FixedScatterers::s_ptr scatterers) {
    m_can_change_cuda_device = false;
    
    const size_t num_scatterers = scatterers->num_scatterers();
    size_t points_common_bytes = num_scatterers*sizeof(float);

    // temporary host memory for scatterer points
    HostPinnedBufferRAII<float> host_temp(points_common_bytes);

    // no point in reallocating if we already have allocated memory and the number of bytes
    // is the same.
    bool reallocate_device_mem = true;
    if (m_device_point_xs && m_device_point_ys && m_device_point_zs && m_device_point_as) {
        if (   (m_device_point_xs->get_num_bytes() == points_common_bytes)
            && (m_device_point_ys->get_num_bytes() == points_common_bytes)
            && (m_device_point_zs->get_num_bytes() == points_common_bytes)
            && (m_device_point_as->get_num_bytes() == points_common_bytes))
        {
            reallocate_device_mem = false;
        }
    }
    if (reallocate_device_mem) {
        m_device_point_xs = std::move(DeviceBufferRAII<float>::u_ptr(new DeviceBufferRAII<float>(points_common_bytes)));
        m_device_point_ys = std::move(DeviceBufferRAII<float>::u_ptr(new DeviceBufferRAII<float>(points_common_bytes)));
        m_device_point_zs = std::move(DeviceBufferRAII<float>::u_ptr(new DeviceBufferRAII<float>(points_common_bytes)));
        m_device_point_as = std::move(DeviceBufferRAII<float>::u_ptr(new DeviceBufferRAII<float>(points_common_bytes)));
    }

    // x values
    for (size_t i = 0; i < num_scatterers; i++) {
        host_temp.data()[i] = scatterers->scatterers[i].pos.x;
    }
    cudaErrorCheck( hipMemcpy(m_device_point_xs->data(), host_temp.data(), points_common_bytes, hipMemcpyHostToDevice) );

    // y values
    for (size_t i = 0; i < num_scatterers; i++) {
        host_temp.data()[i] = scatterers->scatterers[i].pos.y;
    }
    cudaErrorCheck( hipMemcpy(m_device_point_ys->data(), host_temp.data(), points_common_bytes, hipMemcpyHostToDevice) );

    // z values
    for (size_t i = 0; i < num_scatterers; i++) {
        host_temp.data()[i] = scatterers->scatterers[i].pos.z;
    }
    cudaErrorCheck( hipMemcpy(m_device_point_zs->data(), host_temp.data(), points_common_bytes, hipMemcpyHostToDevice) );

    // a values
    for (size_t i = 0; i < num_scatterers; i++) {
        host_temp.data()[i] = scatterers->scatterers[i].amplitude;
    }
    cudaErrorCheck( hipMemcpy(m_device_point_as->data(), host_temp.data(), points_common_bytes, hipMemcpyHostToDevice) );
}

void GpuFixedAlgorithm::set_scatterers(Scatterers::s_ptr new_scatterers) {
    m_can_change_cuda_device = false;
    m_num_scatterers = new_scatterers->num_scatterers();
        
    auto fixed_scatterers = std::dynamic_pointer_cast<FixedScatterers>(new_scatterers);
    if (!fixed_scatterers) {
        throw std::runtime_error("Cast failed");
    }
    copy_scatterers_to_device(fixed_scatterers);
}


}   // end namespace
