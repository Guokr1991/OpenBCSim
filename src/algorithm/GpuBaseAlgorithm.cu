#include "hip/hip_runtime.h"
/*
Copyright (c) 2015, Sigurd Storve
All rights reserved.

Licensed under the BSD license.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of the <organization> nor the
      names of its contributors may be used to endorse or promote products
      derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#include <stdexcept>
#include <iostream>
#include <complex>
#include "GpuBaseAlgorithm.cuh"
#include "gpu_alg_common.cuh"
#include "common_utils.hpp" // for compute_num_rf_samples
#include "discrete_hilbert_mask.hpp"
#include "cuda_debug_utils.h"
#include ""

// Slice a 3D lookup table through plane defined by two unit vectors.
// X- and y-components of grid determines the number of samples.
// NOTE: Number of threads in each block should be one.
__global__ void SliceLookupTable(float3 origin,
                                 float3 dir0,
                                 float3 dir1,
                                 float* output,
                                 hipTextureObject_t lut_tex) {
    const int global_idx = blockIdx.x*gridDim.x + blockIdx.y;
    
    // FORMULA: offset = dim0*num_samples1 + dim1
    const int idx0 = blockIdx.x;  // idx0 = 0..gridDim.x
    const int idx1 = blockIdx.y;  // idx1 = 1..gridDim.y

    // Map to normalized distance in [0.0, 1.0]
    const auto normalized_dist0 = static_cast<float>(idx0)/(gridDim.x-1);
    const auto normalized_dist1 = static_cast<float>(idx1)/(gridDim.y-1);

    const auto tex_pos = origin + dir0*normalized_dist0 + dir1*normalized_dist1;
    output[global_idx] = tex3D<float>(lut_tex, tex_pos.x, tex_pos.y, tex_pos.z);
}

namespace bcsim {
GpuBaseAlgorithm::GpuBaseAlgorithm()
    : m_param_cuda_device_no(0),
      m_can_change_cuda_device(true),
      m_param_num_cuda_streams(2),
      m_num_time_samples(8192),  // TODO: remove this limitation
      m_num_beams_allocated(-1),
      m_param_threads_per_block(128),
      m_store_kernel_details(false),
      m_device_beam_profile(nullptr)
{
    // ensure that CUDA device properties is stored
    save_cuda_device_properties();
}

int GpuBaseAlgorithm::get_num_cuda_devices() const {
    int device_count;
    cudaErrorCheck( hipGetDeviceCount(&device_count) );
    return device_count;
}

void GpuBaseAlgorithm::set_parameter(const std::string& key, const std::string& value) {
    if (key == "gpu_device") {
        if (!m_can_change_cuda_device) {
            throw std::runtime_error("cannot change CUDA device now");            
        }
        const auto device_count = get_num_cuda_devices();
        const int device_no = std::stoi(value);
        if (device_no < 0 || device_no >= device_count) {
            throw std::runtime_error("illegal device number");
        }
        m_param_cuda_device_no = device_no;
        cudaErrorCheck(hipSetDevice(m_param_cuda_device_no));
        save_cuda_device_properties();
    } else if (key == "cuda_streams") {
        const auto num_streams = std::stoi(value);
        if (num_streams <= 0) {
            throw std::runtime_error("invalid number of CUDA streams");
        }
        m_param_num_cuda_streams = num_streams;
    } else if (key == "threads_per_block") {
        const auto threads_per_block = std::stoi(value);
        if (threads_per_block <= 0) {
            throw std::runtime_error("invalid number of threads per block");            
        }
        m_param_threads_per_block = threads_per_block;
    } else if (key == "noise_amplitude") {
        throw std::runtime_error("noise is not yet implemented in GPU algorithms");
    } else if (key == "store_kernel_details") {
        if ((value == "on") || (value == "true")) {
            m_store_kernel_details = true;
        } else if ((value == "off") || (value == "false")) {
            m_store_kernel_details = false;
        } else {
            throw std::runtime_error("invalid value");
        }
    } else {
        BaseAlgorithm::set_parameter(key, value);
    }
}

void GpuBaseAlgorithm::create_cuda_stream_wrappers(int num_streams) {
    m_stream_wrappers.clear();
    for (int i = 0; i < num_streams; i++) {
        m_stream_wrappers.push_back(std::move(CudaStreamRAII::u_ptr(new CudaStreamRAII)));
    }
    m_can_change_cuda_device = false;
}

void GpuBaseAlgorithm::save_cuda_device_properties() {
    const auto num_devices = get_num_cuda_devices();
    if (m_param_cuda_device_no < 0 || m_param_cuda_device_no >= num_devices) {
        throw std::runtime_error("illegal CUDA device number");
    }
    cudaErrorCheck( hipGetDeviceProperties(&m_cur_device_prop, m_param_cuda_device_no) );

    if (m_param_verbose) {
        const auto& p = m_cur_device_prop;
        std::cout << "=== CUDA Device " << m_param_cuda_device_no << ": " << p.name << std::endl;
        std::cout << "Compute capability: "         << p.major << "." << p.minor << std::endl;
        std::cout << "ECCEnabled: "                 << p.ECCEnabled                 << std::endl;
        std::cout << "asyncEngineCount: "           << p.asyncEngineCount           << std::endl;
        std::cout << "canMapHostMemory: "           << p.canMapHostMemory           << std::endl; 
        std::cout << "clockRate: "                  << p.clockRate                  << std::endl;
        std::cout << "computeMode: "                << p.computeMode                << std::endl;
        std::cout << "concurrentKernels: "          << p.concurrentKernels          << std::endl;
        std::cout << "integrated: "                 << p.integrated                 << std::endl;
        std::cout << "kernelExecTimeoutEnabled: "   << p.kernelExecTimeoutEnabled   << std::endl;
        std::cout << "l2CacheSize: "                << p.l2CacheSize                << std::endl;
        std::cout << "maxGridSize: [" << p.maxGridSize[0] << "," << p.maxGridSize[1] << "," << p.maxGridSize[2] << "]\n";
        std::cout << "maxThreadsPerBlock: "         << p.maxThreadsPerBlock         << std::endl;
        std::cout << "memoryBusWidth: "             << p.memoryBusWidth             << std::endl;
        std::cout << "multiProcessorCount: "        << p.multiProcessorCount        << std::endl;
        std::cout << "totalGlobMem: "               << p.totalGlobalMem             << std::endl;
    }
}

void GpuBaseAlgorithm::simulate_lines(std::vector<std::vector<std::complex<bc_float> > >&  /*out*/ rf_lines) {
    m_can_change_cuda_device = false;
    
    if (m_stream_wrappers.size() == 0) {
        create_cuda_stream_wrappers(m_param_num_cuda_streams);
    }

    if (m_store_kernel_details) {
        m_debug_data.clear();
    }
    
    auto num_lines = m_scan_seq->get_num_lines();
    if (num_lines < 1) {
        throw std::runtime_error("No scanlines in scansequence");
    }

    if (!m_beam_profile_configured) {
        throw std::runtime_error("No beam profile is configured");
    }
    
    // compute the number of blocks needed to project all scatterers and check that
    // it is not more than what is supported by the device.
    int num_blocks = round_up_div(m_num_scatterers, m_param_threads_per_block);
    if (num_blocks > m_cur_device_prop.maxGridSize[0]) {
        throw std::runtime_error("required number of x-blocks is larger than device supports");
    }

    // no delay compenasation is needed when returning the projections only
    size_t delay_compensation_num_samples = static_cast<size_t>(m_excitation.center_index);
    const auto num_return_samples = compute_num_rf_samples(m_param_sound_speed, m_scan_seq->line_length, m_excitation.sampling_frequency);
    
    for (int beam_no = 0; beam_no < num_lines; beam_no++) {
        size_t stream_no = beam_no % m_param_num_cuda_streams;
        auto cur_stream = m_stream_wrappers[stream_no]->get();

        std::unique_ptr<EventTimerRAII> event_timer;
        if (m_store_kernel_details) {
            event_timer = std::unique_ptr<EventTimerRAII>(new EventTimerRAII(cur_stream));
            m_debug_data["stream_numbers"].push_back(static_cast<double>(stream_no));
            event_timer->restart();
        }

        if (m_param_verbose) {
            std::cout << "beam_no = " << beam_no << ", stream_no = " << stream_no << std::endl;
        }

        auto scanline = m_scan_seq->get_scanline(beam_no);
        int threads_per_line = 128;
        auto rf_ptr = m_device_time_proj[stream_no]->data();

        // clear time projections (safer than hipMemsetAsync)
        const auto complex_zero = make_hipComplex(0.0f, 0.0f);
        if (m_store_kernel_details) {
            event_timer->restart();
        }
        MemsetKernel<hipComplex><<<m_num_time_samples/threads_per_line, threads_per_line, 0, cur_stream>>>(rf_ptr,
                                                                                                          complex_zero,
                                                                                                          m_num_time_samples);
        if (m_store_kernel_details) {
            const auto elapsed_ms = static_cast<double>(event_timer->stop());
            m_debug_data["kernel_memset_ms"].push_back(elapsed_ms);
            event_timer->restart();
        }

        projection_kernel(stream_no, scanline, num_blocks);
        if (m_store_kernel_details) {
            const auto elapsed_ms = static_cast<double>(event_timer->stop());
            m_debug_data["kernel_projection_ms"].push_back(elapsed_ms);
            event_timer->restart();
        }

        // in-place forward FFT
        cufftErrorCheck( hipfftExecC2C(m_fft_plan->get(), rf_ptr, rf_ptr, HIPFFT_FORWARD) );
        if (m_store_kernel_details) {
            const auto elapsed_ms = static_cast<double>(event_timer->stop());
            m_debug_data["kernel_forward_fft_ms"].push_back(elapsed_ms);
            event_timer->restart();
        }
        
        // multiply with FFT of impulse response w/Hilbert transform
        MultiplyFftKernel<<<m_num_time_samples/threads_per_line, threads_per_line, 0, cur_stream>>>(rf_ptr,
                                                                                                    m_device_excitation_fft->data(),
                                                                                                    m_num_time_samples);
        if (m_store_kernel_details) {
            const auto elapsed_ms = static_cast<double>(event_timer->stop());
            m_debug_data["kernel_multiply_fft_ms"].push_back(elapsed_ms);
            event_timer->restart();
        }

        // in-place inverse FFT
        cufftErrorCheck( hipfftExecC2C(m_fft_plan->get(), rf_ptr, rf_ptr, HIPFFT_BACKWARD) );
        if (m_store_kernel_details) {
            const auto elapsed_ms = static_cast<double>(event_timer->stop());
            m_debug_data["kernel_inverse_fft_ms"].push_back(elapsed_ms);
            event_timer->restart();
        }

        // IQ demodulation (+decimate?)
        const auto f_demod = m_excitation.demod_freq;
        const float norm_f_demod = f_demod/m_excitation.sampling_frequency;
        const float PI = static_cast<float>(4.0*std::atan(1));
        const auto normalized_angular_freq = 2*PI*norm_f_demod;
        DemodulateKernel<<<m_num_time_samples/threads_per_line, threads_per_line, 0, cur_stream>>>(rf_ptr, normalized_angular_freq, m_num_time_samples);
        if (m_store_kernel_details) {
            const auto elapsed_ms = static_cast<double>(event_timer->stop());
            m_debug_data["kernel_demodulate_ms"].push_back(elapsed_ms);
            event_timer->restart();
        }

        // copy to host. Same memory layout?
        const auto num_bytes_iq = sizeof(std::complex<float>)*m_num_time_samples;
        cudaErrorCheck( hipMemcpyAsync(m_host_rf_lines[beam_no]->data(), rf_ptr, num_bytes_iq, hipMemcpyDeviceToHost, cur_stream) ); 
        if (m_store_kernel_details) {
            const auto elapsed_ms = static_cast<double>(event_timer->stop());
            m_debug_data["kernel_memcpy_ms"].push_back(elapsed_ms);
        }
    }
    cudaErrorCheck( hipDeviceSynchronize() );

    // TODO: eliminate unneccessary data copying: it would e.g. be better to
    // only copy what is needed in the above kernel.
    rf_lines.clear();
    for (size_t line_no = 0; line_no < num_lines; line_no++) {
        std::vector<std::complex<bc_float>> temp_samples; // .reserve
        for (size_t i = 0; i < num_return_samples; i += m_radial_decimation) {
            temp_samples.push_back(m_host_rf_lines[line_no]->data()[i+delay_compensation_num_samples]);
        }
        rf_lines.push_back(temp_samples);
    }
}

void GpuBaseAlgorithm::set_excitation(const ExcitationSignal& new_excitation) {
    m_can_change_cuda_device = false;
    
    m_excitation = new_excitation;
    size_t rf_line_bytes   = sizeof(complex)*m_num_time_samples;

    // setup pre-computed convolution kernel and Hilbert transformer.
    m_device_excitation_fft = DeviceBufferRAII<complex>::u_ptr(new DeviceBufferRAII<complex>(rf_line_bytes));
    std::cout << "Number of excitation samples: " << m_excitation.samples.size() << std::endl;
    // convert to complex with zero imaginary part.
    std::vector<std::complex<float> > temp(m_num_time_samples);
    for (size_t i = 0; i < m_excitation.samples.size(); i++) {
        temp[i] = std::complex<float>(m_excitation.samples[i], 0.0f);
    }
    cudaErrorCheck( hipMemcpy(m_device_excitation_fft->data(), temp.data(), rf_line_bytes, hipMemcpyHostToDevice) );
    //dump_device_memory((std::complex<float>*)m_device_excitation_fft.data(), m_num_time_samples, "complex_exitation.txt");

    m_fft_plan = CufftPlanRAII::u_ptr(new CufftPlanRAII(m_num_time_samples, HIPFFT_C2C, 1));

    // compute FFT of excitation signal and add the Hilbert transform
    cufftErrorCheck( hipfftExecC2C(m_fft_plan->get(), m_device_excitation_fft->data(), m_device_excitation_fft->data(), HIPFFT_FORWARD) );
    auto mask = discrete_hilbert_mask<std::complex<float> >(m_num_time_samples);
    DeviceBufferRAII<complex> device_hilbert_mask(rf_line_bytes);
    cudaErrorCheck( hipMemcpy(device_hilbert_mask.data(), mask.data(), rf_line_bytes, hipMemcpyHostToDevice) );
    
    ScaleSignalKernel<<<m_num_time_samples/128, 128>>>(m_device_excitation_fft->data(), 1.0f/m_num_time_samples, m_num_time_samples);
    MultiplyFftKernel<<<m_num_time_samples/128, 128>>>(m_device_excitation_fft->data(), device_hilbert_mask.data(), m_num_time_samples);
    //dump_device_memory((std::complex<float>*) m_device_excitation_fft->data(), m_num_time_samples, "complex_excitation_fft.txt");
}


void GpuBaseAlgorithm::set_scan_sequence(ScanSequence::s_ptr new_scan_sequence) {
    m_can_change_cuda_device = false;
    
    m_scan_seq = new_scan_sequence;

    // HACK: Temporarily limited to the hardcoded value for m_num_time_samples
    auto num_rf_samples = compute_num_rf_samples(m_param_sound_speed, m_scan_seq->line_length, m_excitation.sampling_frequency);
    if (num_rf_samples > m_num_time_samples) {
        std::cout << "num_rf_samples = " << num_rf_samples << std::endl;
        throw std::runtime_error("Too many RF samples required. TODO: remove limitation");
    }

    size_t num_beams = m_scan_seq->get_num_lines();
    // avoid reallocating memory if not necessary.
    if (m_num_beams_allocated < static_cast<int>(num_beams)) {
        std::cout << "Allocating HOST and DEVICE memory: had previously allocated memory for " << m_num_beams_allocated << " beams.\n";
    } else {
        return;
    }

    // allocate host and device memory related to RF lines
    const auto device_iq_line_bytes = sizeof(complex)*m_num_time_samples;
    const auto host_iq_line_bytes   = sizeof(std::complex<float>)*m_num_time_samples;

    m_device_time_proj.resize(m_param_num_cuda_streams);
    for (size_t i = 0; i < m_param_num_cuda_streams; i++) {
        m_device_time_proj[i]    = std::move(DeviceBufferRAII<complex>::u_ptr ( new DeviceBufferRAII<complex>(device_iq_line_bytes)) ); 
    }

    // allocate host memory for all RF lines
    m_host_rf_lines.resize(num_beams);
    for (size_t beam_no = 0; beam_no < num_beams; beam_no++) {
        m_host_rf_lines[beam_no] = std::move(HostPinnedBufferRAII<std::complex<float>>::u_ptr( new HostPinnedBufferRAII<std::complex<float>>(host_iq_line_bytes)) );
    }

    m_num_beams_allocated = static_cast<int>(num_beams);
}

void GpuBaseAlgorithm::set_analytical_profile(IBeamProfile::s_ptr beam_profile) {
    std::cout << "Setting analytical beam profile for GPU algorithm" << std::endl;
    const auto analytical_profile = std::dynamic_pointer_cast<GaussianBeamProfile>(beam_profile);
    if (!analytical_profile) throw std::runtime_error("GpuBaseAlgorithm: failed to cast beam profile");
    m_cur_beam_profile_type = BeamProfile::ANALYTICAL;

    m_analytical_sigma_lat = analytical_profile->getSigmaLateral();
    m_analytical_sigma_ele = analytical_profile->getSigmaElevational();
}

void GpuBaseAlgorithm::set_lookup_profile(IBeamProfile::s_ptr beam_profile) {
    std::cout << "Setting LUT profile for GPU algorithm" << std::endl;
    const auto lut_beam_profile = std::dynamic_pointer_cast<LUTBeamProfile>(beam_profile);
    if (!lut_beam_profile) throw std::runtime_error("GpuBaseAlgorithm: failed to cast beam profile");
    m_cur_beam_profile_type = BeamProfile::LOOKUP;

    int num_samples_rad = lut_beam_profile->getNumSamplesRadial();
    int num_samples_lat = lut_beam_profile->getNumSamplesLateral();
    int num_samples_ele = lut_beam_profile->getNumSamplesElevational();
    std::cout << "=== set_beam_profile() ===" << std::endl;
    std::cout << "num_samples_rad: " << num_samples_rad << std::endl;
    std::cout << "num_samples_lat: " << num_samples_lat << std::endl;
    std::cout << "num_samples_ele: " << num_samples_ele << std::endl;
        
    const auto r_range = lut_beam_profile->getRangeRange();
    const auto l_range = lut_beam_profile->getLateralRange();
    const auto e_range = lut_beam_profile->getElevationalRange();

    // map to linear memory with correct 3D layout
    const auto total = num_samples_rad*num_samples_lat*num_samples_ele;
    std::vector<float> temp_samples;
    temp_samples.reserve(total);
    for (int zi = 0; zi < num_samples_rad; zi++) {
        for (int yi = 0; yi < num_samples_lat; yi++) {
            for (int xi = 0; xi < num_samples_ele; xi++) {
                const auto x = l_range.first + xi*(l_range.last-l_range.first)/(num_samples_lat-1);
                const auto y = e_range.first + yi*(e_range.last-e_range.first)/(num_samples_ele-1);
                const auto z = r_range.first + zi*(r_range.last-r_range.first)/(num_samples_rad-1);
                temp_samples.push_back(lut_beam_profile->sampleProfile(z, x, y));
            }
        }
    }
    m_device_beam_profile = DeviceBeamProfileRAII::u_ptr(new DeviceBeamProfileRAII(DeviceBeamProfileRAII::TableExtent3D(num_samples_lat, num_samples_ele, num_samples_rad),
                                                                                    temp_samples));
    // store spatial extent of profile.
    m_lut_r_min = r_range.first;
    m_lut_r_max = r_range.last;
    m_lut_l_min = l_range.first;
    m_lut_l_max = l_range.last;
    m_lut_e_min = e_range.first;
    m_lut_e_max = e_range.last;

    std::cout << "Created a new DeviceBeamProfileRAII.\n";
    
    // Slice the 3D texture and write as RAW file to disk.    
    const auto write_raw = [&](float3 origin, float3 dir0, float3 dir1, std::string raw_file) {
        const int num_samples = 1024;
        const int total_num_samples = num_samples*num_samples;
        const int num_bytes = sizeof(float)*total_num_samples;
        DeviceBufferRAII<float> device_slice(static_cast<size_t>(num_bytes));
            
        dim3 grid_size(num_samples, num_samples, 1);
        dim3 block_size(1, 1, 1);
        SliceLookupTable<<<grid_size, block_size>>>(origin, dir0, dir1,
                                                    device_slice.data(),
                                                    m_device_beam_profile->get());
        cudaErrorCheck( hipDeviceSynchronize() );
        dump_device_buffer_as_raw_file(device_slice, raw_file);
    };

    const std::string raw_path("d:/temp/raw_lookup_table/");
    // slice in the middle lateral-elevational plane (radial dist is 0.5)
    write_raw(make_float3(0.0f, 0.0f, 0.5f),
                make_float3(1.0f, 0.0f, 0.0f),
                make_float3(0.0f, 1.0f, 0.0f),
                raw_path + "lut_slice_lat_ele.raw");
    // slice the middle lateral-radial plane (elevational dist is 0.5)
    write_raw(make_float3(0.0f, 0.5f, 0.0f),
                make_float3(1.0f, 0.0f, 0.0f),
                make_float3(0.0f, 0.0f, 1.0f),
                raw_path + "lut_slice_lat_rad.raw");
    // slice the middle elevational-radial plane (lateral dist is 0.5)
    write_raw(make_float3(0.5f, 0.0f, 0.0f),
                make_float3(0.0f, 1.0f, 0.0f),
                make_float3(0.0f, 0.0f, 1.0f),
                raw_path + "lut_slice_ele_rad.raw");
}

}   // end namespace

