#include "cuda_kernels_common.cuh"

template <typename T>
void launch_MemsetKernel(int grid_size, int block_size, hipStream_t stream, T* ptr, T value, int num_samples) {
    MemsetKernel<hipComplex><<<grid_size, block_size, 0, stream>>>(ptr, value, num_samples);
}

void launch_MultiplyFftKernel(int grid_size, int block_size, hipStream_t stream, hipfftComplex* time_proj_fft, const hipfftComplex* filter_fft, int num_samples) {
    MultiplyFftKernel<<<grid_size, block_size, 0, stream>>>(time_proj_fft, filter_fft, num_samples);
}

// explicit function template instantiations for required datatypes
template void launch_MemsetKernel(int grid_size, int block_size, hipStream_t stream, hipComplex* ptr, hipComplex value, int num_samples);
