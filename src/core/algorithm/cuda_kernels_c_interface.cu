#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include "cuda_kernels_common.cuh"

template <typename T>
void launch_MemsetKernel(int grid_size, int block_size, hipStream_t stream, T* ptr, T value, int num_samples) {
    MemsetKernel<hipComplex><<<grid_size, block_size, 0, stream>>>(ptr, value, num_samples);
}

template void launch_MemsetKernel(int grid_size, int block_size, hipStream_t stream, hipComplex* ptr, hipComplex value, int num_samples);
